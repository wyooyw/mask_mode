#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#define CEIL(x,n) ((x+n-1)/n)

template <typename scalar_t>
__global__ void mask_mode_cuda_kernel(
    scalar_t* tensor_in, // row * column
    scalar_t* tensor_mask, // row * column
    int* tensor_out, // row
    int row, // tensor_in的行数
    int column, // tensor_in的列数
    int ele_num_per_thread // 每个线程处理的元素数量
    ) {
      __shared__ int counts[1024 * 9];
      int block_idx = blockIdx.x;
      int thread_idx = threadIdx.x;

      int input_row_idx = block_idx; // 当前block处理的行
      int input_begin_addr = input_row_idx * column;
      int input_col_idx_begin = thread_idx * ele_num_per_thread; // 当前thread处理的起始列
      int input_col_idx_end = min(input_col_idx_begin + ele_num_per_thread, column); // 当前thread处理的终止列(不包含)

      int counts_begin_addr = thread_idx * 9;
      int output_begin_addr = block_idx;

      int activate_thread_num = CEIL(column,ele_num_per_thread);
      int mode_val = -1;
      int mode_cnt = -1;

      if(input_row_idx < row){
        if(input_col_idx_begin < column){

          // 清零
          for(int i = 0; i < 9;i++){
            counts[counts_begin_addr + i] = 0;
          }

          // 线程内计数
          for(int i = input_col_idx_begin; i<input_col_idx_end;i++){
            scalar_t ele = tensor_in[input_begin_addr + i];
            counts[counts_begin_addr + ele] += (1 & tensor_mask[input_begin_addr + i]);
          }
          
        }

        // 线程间累加
        int acc_thread_num = activate_thread_num / 2;
        int acc_thread_num_ceil = CEIL(activate_thread_num, 2);
        do{
          
          if(thread_idx < acc_thread_num){
            for(int i = 0 ; i < 9; i++){
              counts[thread_idx * 9 + i] += counts[(thread_idx + acc_thread_num_ceil) * 9 + i];
            }
          }
           __syncthreads();
          acc_thread_num = acc_thread_num_ceil / 2 ;
          acc_thread_num_ceil = CEIL(acc_thread_num_ceil, 2);
        }while(acc_thread_num > 0);
        
        __syncthreads();

        // 将counts拷贝到tensor_out
        if(thread_idx==0){
          for(int i = 0 ; i < 9 ; i++){
            if(counts[i] > mode_cnt){
              mode_cnt = counts[i];
              mode_val = i;
            }
          }
          tensor_out[output_begin_addr] = mode_val;
        }
      }

}

torch::Tensor mask_mode_cuda(
    torch::Tensor tensor_in, torch::Tensor tensor_mask,int grid_size, int block_size) {
  int row = tensor_in.size(0);
  int column = tensor_in.size(1);

  int ele_num_per_thread = CEIL(column, block_size);;
  

  auto tensor_out = torch::zeros({row},
                     torch::dtype(torch::kInt).device(tensor_in.device()));

  AT_DISPATCH_INTEGRAL_TYPES(tensor_in.type(), "mask_mode_cuda", ([&] {
    mask_mode_cuda_kernel<scalar_t><<<grid_size, block_size>>>(
        tensor_in.data<scalar_t>(),
        tensor_mask.data<scalar_t>(),
        tensor_out.data<int>(),
        row,
        column,
        ele_num_per_thread
        );
  }));
 
  return tensor_out;
}


template <typename scalar_t>
__global__ void mode_cuda_kernel(
    scalar_t* tensor_in, // row * column
    int* tensor_out, // row
    int row, // tensor_in的行数
    int column, // tensor_in的列数
    int ele_num_per_thread // 每个线程处理的元素数量
    ) {
      __shared__ int counts[1024 * 9];
      int block_idx = blockIdx.x;
      int thread_idx = threadIdx.x;

      int input_row_idx = block_idx; // 当前block处理的行
      int input_begin_addr = input_row_idx * column;
      int input_col_idx_begin = thread_idx * ele_num_per_thread; // 当前thread处理的起始列
      int input_col_idx_end = min(input_col_idx_begin + ele_num_per_thread, column); // 当前thread处理的终止列(不包含)

      int counts_begin_addr = thread_idx * 9;
      int output_begin_addr = block_idx;

      int activate_thread_num = CEIL(column,ele_num_per_thread);
      int mode_val = -1;
      int mode_cnt = -1;

      if(input_row_idx < row){
        if(input_col_idx_begin < column){

          // 清零
          for(int i = 0; i < 9;i++){
            counts[counts_begin_addr + i] = 0;
          }

          // 线程内计数
          for(int i = input_col_idx_begin; i<input_col_idx_end;i++){
            scalar_t ele = tensor_in[input_begin_addr + i];
            counts[counts_begin_addr + ele] += 1;
          }
          
        }

        // 线程间累加
        int acc_thread_num = activate_thread_num / 2;
        int acc_thread_num_ceil = CEIL(activate_thread_num, 2);
        do{
          
          if(thread_idx < acc_thread_num){
            for(int i = 0 ; i < 9; i++){
              counts[thread_idx * 9 + i] += counts[(thread_idx + acc_thread_num_ceil) * 9 + i];
            }
          }
           __syncthreads();
          acc_thread_num = acc_thread_num_ceil / 2 ;
          acc_thread_num_ceil = CEIL(acc_thread_num_ceil, 2);
        }while(acc_thread_num > 0);
        
        __syncthreads();

        // 将counts拷贝到tensor_out
        if(thread_idx==0){
          for(int i = 0 ; i < 9 ; i++){
            if(counts[i] > mode_cnt){
              mode_cnt = counts[i];
              mode_val = i;
            }
          }
          tensor_out[output_begin_addr] = mode_val;
        }
      }

}

torch::Tensor mode_cuda(
    torch::Tensor tensor_in, int grid_size, int block_size) {
  int row = tensor_in.size(0);
  int column = tensor_in.size(1);

  int ele_num_per_thread = CEIL(column, block_size);;
  

  auto tensor_out = torch::zeros({row},
                     torch::dtype(torch::kInt).device(tensor_in.device()));

  AT_DISPATCH_INTEGRAL_TYPES(tensor_in.type(), "mode_cuda", ([&] {
    mode_cuda_kernel<scalar_t><<<grid_size, block_size>>>(
        tensor_in.data<scalar_t>(),
        tensor_out.data<int>(),
        row,
        column,
        ele_num_per_thread
        );
  }));
 
  return tensor_out;
}